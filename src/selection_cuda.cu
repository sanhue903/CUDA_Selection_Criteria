#include "hip/hip_runtime.h"
// selection_cuda.cu

#include <hip/hip_runtime.h>
#include <cstdint>
#include "include/criteria_sketch_cuda.cuh"

// === kernel 1: solo smh_a ===============================================
__global__ void kernel_smh(const uint64_t* sketches,
                           const double* cards,
                           int N, int m,
                           int n_rows, int n_bands,
                           double tau,
                           int* out)
{
    int total_pairs = N*(N-1)/2;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_pairs) return;

    // map idx -> (i, k)
    int i = N - 2 - int(sqrtf(-8*idx + 4*N*(N-1)-7)*0.5f - 0.5f);
    int k = idx + i + 1 - N*(N-i)/2 + (N-i)*((N-i)-1)/2;

    const uint64_t* v1 = sketches + i*m;
    const uint64_t* v2 = sketches + k*m;

    out[idx] = smh_a(v1, v2, n_rows, n_bands);
}

// === kernel 2: CB + smh_a  ==============================================
__global__ void kernel_CBsmh(const uint64_t* sketches,
                             const double* cards,
                             int N, int m,
                             int n_rows, int n_bands,
                             double tau,
                             int* out)
{
    int total_pairs = N*(N-1)/2;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_pairs) return;

    int i = N - 2 - int(sqrtf(-8*idx + 4*N*(N-1)-7)*0.5f - 0.5f);
    int k = idx + i + 1 - N*(N-i)/2 + (N-i)*((N-i)-1)/2;

    double e1 = cards[i];
    double e2 = cards[k];
    const uint64_t* v1 = sketches + i*m;
    const uint64_t* v2 = sketches + k*m;
    out[idx] = CB(tau, e1, e2) || smh_a(v1, v2, n_rows, n_bands);
}

// === WRAPPERS ===========================================================

void launch_kernel_smh(const uint64_t* d_sketches,
                                  const double* d_cards,
                                  int N, int m,
                                  int n_rows, int n_bands,
                                  double tau,
                                  int* d_out,
                                  int blockSize,
                                  int gridSize)
{
    kernel_smh<<<gridSize, blockSize>>>(d_sketches, d_cards, N, m, n_rows, n_bands, tau, d_out);
    hipDeviceSynchronize(); // Optionally check errors!
}

void launch_kernel_CBsmh(const uint64_t* d_sketches,
                                    const double* d_cards,
                                    int N, int m,
                                    int n_rows, int n_bands,
                                    double tau,
                                    int* d_out,
                                    int blockSize,
                                    int gridSize)
{
    kernel_CBsmh<<<gridSize, blockSize>>>(d_sketches, d_cards, N, m, n_rows, n_bands, tau, d_out);
    hipDeviceSynchronize(); // Optionally check errors!
}
